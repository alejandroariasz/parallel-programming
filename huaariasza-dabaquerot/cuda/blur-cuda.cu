#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#include <cv.h>
#include <cxcore.h>
#include <highgui.h>

#include <ctime>
#include <cstdlib>
#include <iostream>

using namespace cv;
using namespace std;

int threads;
int kernel_size;
int indexes_count;
int blocks_per_grid;

//function to convert to string 
template <typename T>
string Str(const T & t)
{
	ostringstream os;
	os << t;
	return os.str();
}

/*   function to safely manage error un cuda memory allocation   */
static inline void _safe_cuda_call(hipError_t err, const char* msg, const char* file_name, const int line_number)
{
	if(err != hipSuccess)
	{
		fprintf(stderr,"%s\n\nFile: %s\n\nLine Number: %d\n\nReason: %s\n", msg,file_name, line_number, hipGetErrorString(err));
		std::cin.get();
		exit(EXIT_FAILURE);
	}
}
#define SAFE_CALL(call,msg) _safe_cuda_call((call),(msg),__FILE__,__LINE__)


/*   function to compute blur   */
__global__ void blur_img_kernel(short int* dRed, short int* dGreen, short int* dBlue, bool* dEdited, int cols, int rows, int kernel, int blockSize)
{
	//Thread index	
	const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;

	//variables to allocate the edges of the working thread
	int startBlockX = xIndex * blockSize;
	int endBlockX = (xIndex + 1) * blockSize;
	
	//variables to allocate the edges of the kernel
	int startX = 0;
	int startY = 0;
	int endX = cols;
	int endY = rows;

	//Index to modify	
	int vectIndex = 0;

	//variable to iterate over the kernel indexes
	int subindex = 0;

	//variables to compute pixel sum inside the kernel (Red, Green, Blue)
	double averageR = 0;
	double averageG = 0;
	double averageB = 0;

	//cumpute the total of pixels (technically kernel²)
	double total = 0; 

	//handling error on overflow because of the last block exceeds number of cols	
	if(endBlockX > cols)
		endBlockX = cols;

	//iterate over the block of the working thread
	for(int i = startBlockX; i < endBlockX; i++)
	{
		for(int j = 0; j < rows; j++)
		{
			total = 0;				
			subindex = 0;
			vectIndex = i + (j * cols);

			averageR = 0;
			averageG = 0;
			averageB = 0;

			if(dEdited[vectIndex] == true) continue; //control if the pixel was modified by another thread

			//Region compute kernel edges

			startX = 0;
			if(i - (kernel - 1) / 2  > 0)
				startX = i - (kernel - 1) / 2;

			startY = 0;
			if(j - (kernel - 1) / 2  > 0)
				startY = j - (kernel - 1) / 2;

			endX = cols;
			if(i + (kernel - 1) / 2  < cols)
				endX = i + (kernel - 1) / 2;

			endY = rows;
			if(j + (kernel - 1) / 2  < rows)
				endY = j + (kernel - 1) / 2;
			
			//End Region

			//if kernel is equal to 1 return the pixel as it is
			if(kernel == 1)
			{
				averageR = dRed[vectIndex];
				averageG = dGreen[vectIndex];
				averageB = dBlue[vectIndex];
				total = 1;
			}

			//iterate over the kernel computing pixel RGB sums and total of pixels
			for(int k = startX; k <= endX; k++)
			{
				for(int k2 = startY; k2 <= endY; k2++)
				{
				    	subindex = k + (k2 * cols);
					averageR = averageR + (double)dRed[subindex];
			    		averageG = averageG + (double)dGreen[subindex];
			    		averageB = averageB + (double)dBlue[subindex];
					total = total + 1;
				}
			}

			//assign average value per channel of color (RGB)
			dRed[vectIndex] = (short int)(averageR / total);
			dGreen[vectIndex] = (short int)(averageG / total);
			dBlue[vectIndex] = (short int)(averageB / total);
			
			//flag to check the current pixel as modified
			dEdited[vectIndex] = true;
		}
	}
}

/*   function to allocate memory, call cuda blur fuction and apply to output the blur effect   */
void make_blur(const cv::Mat& input, cv::Mat& output)
{
	//input variables	
	int rows = input.rows;
	int cols = input.cols;
	int kernel = kernel_size;

	//size of the image (number of pixels
	int imgSize = input.rows * input.cols;

	//number of parallel tasks
	int indexes = blocks_per_grid * threads;

	//size of the block taken of the image per task
	int blockSize = ceil((double)cols / (double)indexes);

	//variables to store RGB pixels in host memory
	short int *h_red = new short int[imgSize];
	short int *h_green = new short int[imgSize];
	short int *h_blue = new short int[imgSize];

	//variables to store RGB pixels in device memory
     	short int *d_red, *d_green, *d_blue;

	//variable to know if a pixel was modified in host memory
	bool *h_edited = new bool[imgSize];

	//variable to know if a pixel was modified in device memory
	bool *d_edited;

	Mat inputCopy = input.clone();

	//size of a pixel color array
	int colorSize = sizeof(short int) * imgSize; 

	//size of a boolean array of the image
	int editedSize = sizeof(bool) * imgSize;

	//iterate over the image pixel to initialize the data of host memory
	int index = 0;
	for(int i = 0; i < cols; i++)
    	{		
        	for(int j = 0; j < rows; j++)
        	{
			index = i + (j * cols);
			Vec3b vect = inputCopy.at<Vec3b>(Point(i, j));
		    	h_red[index] = (short int)vect[0];
		    	h_green[index] = (short int)vect[1];
		        h_blue[index] = (short int)vect[2];
			h_edited[index] = false;
        	}
    	}

	inputCopy.release();
     
	//Region allocate host memory data in device memory

	SAFE_CALL(hipMalloc<short int>(&d_red, colorSize), "CUDA Malloc Failed");
	SAFE_CALL(hipMalloc<short int>(&d_green, colorSize), "CUDA Malloc Failed");
	SAFE_CALL(hipMalloc<short int>(&d_blue, colorSize), "CUDA Malloc Failed");
	SAFE_CALL(hipMalloc<bool>(&d_edited, editedSize), "CUDA Malloc Failed");

	//End Region

	//Region copy data from OpenCV input image to device memory

	SAFE_CALL(hipMemcpy(d_red, h_red, colorSize, hipMemcpyHostToDevice), "CUDA Memcpy Host To Device Failed");	
	SAFE_CALL(hipMemcpy(d_green, h_green, colorSize, hipMemcpyHostToDevice), "CUDA Memcpy Host To Device Failed");
	SAFE_CALL(hipMemcpy(d_blue, h_blue, colorSize, hipMemcpyHostToDevice), "CUDA Memcpy Host To Device Failed");
	SAFE_CALL(hipMemcpy(d_edited, h_edited, editedSize, hipMemcpyHostToDevice), "CUDA Memcpy Host To Device Failed");

	//End Region

	int threads_per_block = threads / blocks_per_grid;

	//launch the blur conversion kernel
	blur_img_kernel<<<blocks_per_grid, threads_per_block>>>(d_red, d_green, d_blue, d_edited, cols, rows, kernel, blockSize);

	//synchronize tu check errors in any kernel
	SAFE_CALL(hipDeviceSynchronize(), "Kernel Launch Failed");

	//Region retrieve memory from device to host

	SAFE_CALL(hipMemcpy(h_red, d_red, colorSize, hipMemcpyDeviceToHost), "CUDA Memcpy Host To Device Failed");
	SAFE_CALL(hipMemcpy(h_green, d_green, colorSize, hipMemcpyDeviceToHost), "CUDA Memcpy Host To Device Failed");
	SAFE_CALL(hipMemcpy(h_blue, d_blue, colorSize, hipMemcpyDeviceToHost), "CUDA Memcpy Host To Device Failed");

	//End Region

	//Region free the device memory

	SAFE_CALL(hipFree(d_red), "CUDA Free Failed");
	SAFE_CALL(hipFree(d_green), "CUDA Free Failed");
	SAFE_CALL(hipFree(d_blue), "CUDA Free Failed");
	SAFE_CALL(hipFree(d_edited), "CUDA Free Failed");

	//End Region

	//Region create output image

	Vec3d outVect;
	int index2 = 0;

	//iterate over image size to assign pixel color results in output
    	for(int io = 0; io < cols; io++)
	{
		for(int jo = 0; jo < rows; jo++)
		{
			index2 = io + (jo * cols);
			outVect = Vec3d((double)h_red[index2]/255.0, (double)h_green[index2]/255.0, (double)h_blue[index2]/255.0);
			output.at<Vec3d>(Point(io, jo)) = outVect;
		}
	}

	//End Region

	//Region free host memory
	
	free(h_red);
	free(h_green);
	free(h_blue);
	free(h_edited);

	//End Region
}

int main(int argc, char** argv)
{
	//if there was an error with the input parameters
	if(argc != 5)
	{
		cout << "Missing or incorrect input parameters" << endl;
		cout << "Params:" << endl;
		cout << "1. image name: example mario.jpg" << endl;
		cout << "2. kernel: size odd number example: 17" << endl;
		cout << "3. threads: number of gpu threads that will be used example: 192";
		cout << "4. block2: number of gpu blocks that will be used example: 2";
		cout << "5. is testing: 0 to display images 1 to enable testing mode";
		return 0;
	}

	//variable to store input image name
	char* image_name = (char *)malloc(sizeof(char) * 256);

	//flag to set testing mode(not display image)
	int isTesting = 0;

	//Region capture input params

	sscanf(argv[1], "%s", image_name);
	sscanf(argv[2], "%i", &kernel_size);
	sscanf(argv[3], "%i", &threads);	
	sscanf(argv[4], "%i", &blocks_per_grid);
	sscanf(argv[5], "%i", &isTesting);

	//End Region

	//variable to store start time
	int start_s = clock();

	//Region reading input image in img folder

	Mat input = imread(Str("img/") + image_name, 1);
	if (input.empty())
	{
		cout << "error: image not read from file\n\n";
		return(0);
	}

	//End Region

	//validate that kernel is odd
	if (kernel_size % 2 == 0)
	{
		cout << "error: arg 2 kernel size must be odd\n\n";
		return(0);
	}

	//validate that number of threads is divisible on the number of blocks
	if (threads % blocks_per_grid != 0)
	{
		cout << Str(threads % blocks_per_grid) + " error: args 3 and 4 number of threads(3) must be divisible in blocks per grid(4)\n\n";
		return(0);
	}

	//width and height of the image
	int rows = input.rows;
	int cols = input.cols;

	//create Mat for output image
	cv::Mat output(rows, cols, CV_64FC3);

	//launch function to apply blur
	make_blur(input, output);

	//if testing mode is disabled then show images
	if(isTesting == 0)
	{
		namedWindow("Input", CV_WINDOW_NORMAL);
		namedWindow("Output", CV_WINDOW_NORMAL);
		cv::imshow("Input", input);
		cv::imshow("Output",output);
		cv::waitKey();
	}

	//variable to store end time
	int stop_s = clock();

	//free images memory
	input.release();
	output.release();
	
	//print performance information
	cout << cols << "x";
	cout << rows << "\t";
	cout << threads << "\t";
	cout << blocks_per_grid << "\t";
	cout << kernel_size << "\t";
    	cout << (stop_s-start_s)/double(CLOCKS_PER_SEC)*1000 << " ms" << endl;

	return 0;
}
