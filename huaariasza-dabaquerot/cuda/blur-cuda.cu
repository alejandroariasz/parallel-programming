#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#include <cv.h>
#include <cxcore.h>
#include <highgui.h>

#include <ctime>
#include <cstdlib>
#include <iostream>

using namespace cv;
using namespace std;

int threads;
int kernel_size;
int indexes_count;
int blocks_per_grid;

template <typename T>
string Str(const T & t)
{
	ostringstream os;
	os << t;
	return os.str();
}


static inline void _safe_cuda_call(hipError_t err, const char* msg, const char* file_name, const int line_number)
{
	if(err != hipSuccess)
	{
		fprintf(stderr,"%s\n\nFile: %s\n\nLine Number: %d\n\nReason: %s\n", msg,file_name, line_number, hipGetErrorString(err));
		std::cin.get();
		exit(EXIT_FAILURE);
	}
}

#define SAFE_CALL(call,msg) _safe_cuda_call((call),(msg),__FILE__,__LINE__)

__global__ void blur_img_kernel(short int* dRed, short int* dGreen, short int* dBlue, bool* dEdited, int cols, int rows, int kernel, int blockSize)
{	
	const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	
	int startX = 0;
	int startY = 0;
	int endX = cols;
	int endY = rows;

	double total = 0;				
	int subindex = 0;
	int vectIndex = 0;

	double averageR = 0;
	double averageG = 0;
	double averageB = 0;

	int startBlockX = xIndex * blockSize;
	int endBlockX = (xIndex + 1) * blockSize;

	if(endBlockX > cols)
		endBlockX = cols;

	for(int i = startBlockX; i < endBlockX; i++)
	{
		if(i >= cols) continue;

		for(int j = 0; j < rows; j++)
		{
			total = 0;				
			subindex = 0;
			vectIndex = i + (j * cols);

			averageR = 0;
			averageG = 0;
			averageB = 0;

			if(dEdited[vectIndex] == true) continue;

			startX = 0;
			if(i - (kernel - 1) / 2  > 0)
				startX = i - (kernel - 1) / 2;

			startY = 0;
			if(j - (kernel - 1) / 2  > 0)
				startY = j - (kernel - 1) / 2;

			endX = cols;
			if(i + (kernel - 1) / 2  < cols)
				endX = i + (kernel - 1) / 2;

			endY = rows;
			if(j + (kernel - 1) / 2  < rows)
				endY = j + (kernel - 1) / 2;

			if(kernel == 1)
			{
				averageR = dRed[vectIndex];
				averageG = dGreen[vectIndex];
				averageB = dBlue[vectIndex];
				total = 1;
			}

			for(int k = startX; k <= endX; k++)
			{
				for(int k2 = startY; k2 <= endY; k2++)
				{
				    	subindex = k + (k2 * cols);
					averageR = averageR + (double)dRed[subindex];
			    		averageG = averageG + (double)dGreen[subindex];
			    		averageB = averageB + (double)dBlue[subindex];
					total = total + 1;
				}
			}

			dRed[vectIndex] = (short int)(averageR / total);
			dGreen[vectIndex] = (short int)(averageG / total);
			dBlue[vectIndex] = (short int)(averageB / total);
			dEdited[vectIndex] = true;
		}
	}
}


void make_blur(const cv::Mat& input, cv::Mat& output)
{	
	int rows = input.rows;
	int cols = input.cols;
	int kernel = kernel_size;
	int imgSize = input.rows * input.cols;
	int indexes = blocks_per_grid * threads;
	int blockSize = ceil((double)cols / (double)indexes);


	short int *h_red = new short int[imgSize];
	short int *h_green = new short int[imgSize];
	short int *h_blue = new short int[imgSize];

     	short int *d_red, *d_green, *d_blue;

	bool *h_edited = new bool[imgSize];

	bool *d_edited;

	Mat inputCopy = input.clone();

	int colorSize = sizeof(short int) * imgSize; 
	int editedSize = sizeof(bool) * imgSize;

	int index = 0;
	for(int i = 0; i < cols; i++)
    	{		
        	for(int j = 0; j < rows; j++)
        	{
			index = i + (j * cols);
			Vec3b vect = inputCopy.at<Vec3b>(Point(i, j));
		    	h_red[index] = (short int)vect[0];
		    	h_green[index] = (short int)vect[1];
		        h_blue[index] = (short int)vect[2];
			h_edited[index] = false;
        	}
    	}

	inputCopy.release();
     
	SAFE_CALL(hipMalloc<short int>(&d_red, colorSize), "CUDA Malloc Failed");
	SAFE_CALL(hipMalloc<short int>(&d_green, colorSize), "CUDA Malloc Failed");
	SAFE_CALL(hipMalloc<short int>(&d_blue, colorSize), "CUDA Malloc Failed");
	SAFE_CALL(hipMalloc<bool>(&d_edited, editedSize), "CUDA Malloc Failed");

	//Copy data from OpenCV input image to device memory
	SAFE_CALL(hipMemcpy(d_red, h_red, colorSize, hipMemcpyHostToDevice), "CUDA Memcpy Host To Device Failed");	
	SAFE_CALL(hipMemcpy(d_green, h_green, colorSize, hipMemcpyHostToDevice), "CUDA Memcpy Host To Device Failed");
	SAFE_CALL(hipMemcpy(d_blue, h_blue, colorSize, hipMemcpyHostToDevice), "CUDA Memcpy Host To Device Failed");
	SAFE_CALL(hipMemcpy(d_edited, h_edited, editedSize, hipMemcpyHostToDevice), "CUDA Memcpy Host To Device Failed");

	//Launch the blur conversion kernel
	blur_img_kernel<<<blocks_per_grid, threads>>>(d_red, d_green, d_blue, d_edited, cols, rows, kernel, blockSize);

	//Synchronize to check for any kernel launch errors
	SAFE_CALL(hipDeviceSynchronize(), "Kernel Launch Failed");

	//Copy back data from destination device meory to OpenCV output image
	SAFE_CALL(hipMemcpy(h_red, d_red, colorSize, hipMemcpyDeviceToHost), "CUDA Memcpy Host To Device Failed");
	SAFE_CALL(hipMemcpy(h_green, d_green, colorSize, hipMemcpyDeviceToHost), "CUDA Memcpy Host To Device Failed");
	SAFE_CALL(hipMemcpy(h_blue, d_blue, colorSize, hipMemcpyDeviceToHost), "CUDA Memcpy Host To Device Failed");

	//Free the device memory
	SAFE_CALL(hipFree(d_red), "CUDA Free Failed");
	SAFE_CALL(hipFree(d_green), "CUDA Free Failed");
	SAFE_CALL(hipFree(d_blue), "CUDA Free Failed");
	SAFE_CALL(hipFree(d_edited), "CUDA Free Failed");

	Mat outputCopy = output.clone();

	Vec3d outVect;
	int index2 = 0;

    	for(int io = 0; io < cols; io++)
	{
		for(int jo = 0; jo < rows; jo++)
		{
			index2 = io + (jo * cols);
			outVect = Vec3d((double)h_red[index2]/255.0, (double)h_green[index2]/255.0, (double)h_blue[index2]/255.0);
			output.at<Vec3d>(Point(io, jo)) = outVect;
		}
	}

	free(h_red);
	free(h_green);
	free(h_blue);
	free(h_edited);
}

int main(int argc, char** argv)
{
	char* image_name;
	image_name = (char *)malloc(sizeof(char) * 256);

	int isTesting = 0;

	sscanf(argv[1], "%s", image_name);
	sscanf(argv[2], "%i", &kernel_size);
	sscanf(argv[3], "%i", &threads);	
	sscanf(argv[4], "%i", &blocks_per_grid);
	sscanf(argv[5], "%i", &isTesting);

	int start_s = clock();

	Mat input = imread(Str("img/") + image_name, 1);
	if (input.empty())
	{
		cout << "error: image not read from file\n\n";
		return(0);
	}

	if (kernel_size % 2 == 0)
	{
		cout << "error: arg 2 kernel size must be odd\n\n";
		return(0);
	}

	if (threads % blocks_per_grid != 0)
	{
		cout << Str(threads % blocks_per_grid) + " error: args 3 and 4 number of threads(3) must be divisible in blocks per grid(4)\n\n";
		return(0);
	}

	int rows = input.rows;
	int cols = input.cols;

	cv::Mat output(rows, cols, CV_64FC3);
	make_blur(input, output);

	if(isTesting == 0)
	{
		namedWindow("Input", CV_WINDOW_NORMAL);
		namedWindow("Output", CV_WINDOW_NORMAL);
		cv::imshow("Input", input);
		cv::imshow("Output",output);
		cv::waitKey();
	}
	
	int stop_s = clock();

	input.release();
	output.release();
	
	cout << cols << "x";
	cout << rows << "\t";
	cout << threads << "\t";
	cout << blocks_per_grid << "\t";
	cout << kernel_size << "\t";
    	cout << (stop_s-start_s)/double(CLOCKS_PER_SEC)*1000 << " ms" << endl;

	return 0;
}
